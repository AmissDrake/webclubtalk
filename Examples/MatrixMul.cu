#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define BLOCK_SIZE 32

__global__ void matrixMultiplyKernel(int *A, int *B, int *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if (row < N && col < N) {
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

void initializeMatrix(int *matrix, int N) {
    for (int i = 0; i < N * N; i++) {
        matrix[i] = rand() % 10;
    }
}

int main() {
    int N = 1900;
    size_t size = N * N * sizeof(int);
    int *h_A, *h_B, *h_C;
    int *d_A, *d_B, *d_C;
    hipEvent_t start, stop;
    float gpu_time = 0.0f;

    // Allocate host memory
    h_A = (int*)malloc(size);
    h_B = (int*)malloc(size);
    h_C = (int*)malloc(size);

    // Initialize matrices
    srand(time(NULL));
    initializeMatrix(h_A, N);
    initializeMatrix(h_B, N);

    // Allocate device memory
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Set up timing
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // Launch kernel
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y);
    matrixMultiplyKernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Record time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("Matrix multiplication of 2 %dx%d matrices took %f seconds.\n", N, N, gpu_time / 1000.0f);

    // Free memory
    free(h_A); free(h_B); free(h_C);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    hipEventDestroy(start); hipEventDestroy(stop);

    return 0;
}
